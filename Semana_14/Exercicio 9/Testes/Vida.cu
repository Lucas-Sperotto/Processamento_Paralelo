#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>

// prototipo de wall_time
double wall_time(void);


// UmaVida: Executa uma iteracao do Jogo da Vida
//          em tabuleiros de tamanho tam. Produz o tabuleiro
//          de saida tabulOut a partir do tabuleiro de entrada
//          tabulIn. Os tabuleiros tem tam x tam celulas
//          internas vivas ou mortas. O tabuleiro eh orlado 
//          por celulas eternamente mortas.
//          O tabuleiro eh dimensionado tam+2 x tam+2.

void UmaVida(int* tabulIn_h, int* tabulOut_h, int tam) {
  int i, j, jAnt, jCor, jPro;
  int vizviv;
  
  for (j=1; j<=tam; j++) {
    jAnt = (j-1)*(tam+2);  // inicio linha anterior
    jCor =  j   *(tam+2);  // inicio linha corrente
    jPro = (j+1)*(tam+2);  // inicio linha proxima
    for (i= 1; i<=tam; i++) {
      vizviv = 
	tabulIn_h[i-1 + jAnt] + 
	tabulIn_h[i   + jAnt] +
	tabulIn_h[i+1 + jAnt] + 
	tabulIn_h[i-1 + jCor] + 
	tabulIn_h[i+1 + jCor] + 
	tabulIn_h[i-1 + jPro] + 
	tabulIn_h[i   + jPro] + 
	tabulIn_h[i+1 + jPro];
      if (tabulIn_h[i + jCor] && vizviv < 2) 
	tabulOut_h[i + jCor] = 0;
      else if (tabulIn_h[i + jCor] && vizviv > 3) 
	tabulOut_h[i + jCor] = 0;
      else if (!tabulIn_h[i + jCor] && vizviv == 3) 
	tabulOut_h[i + jCor] = 1;
      else
	tabulOut_h[i + jCor] = tabulIn_h[i + jCor];
    }
  }
}


// Kernel

__global__ void UmaVidaGPU(int* tabulIn_d, int* tabulOut_d, int tam) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1; 
  int jAnt, jCor, jPro;
  int vizviv;
  
  //for (j=1; j<=tam; j++){ 
  if (j <= tam){
    jAnt = (j-1)*(tam+2);  // inicio linha anterior
    jCor =  j   *(tam+2);  // inicio linha corrente
    jPro = (j+1)*(tam+2);  // inicio linha proxima
    //for (i= 1; i<=tam; i++) {
	if (i <= tam){
      vizviv = 
	tabulIn_d[i-1 + jAnt] + 
	tabulIn_d[i   + jAnt] +
	tabulIn_d[i+1 + jAnt] + 
	tabulIn_d[i-1 + jCor] + 
	tabulIn_d[i+1 + jCor] + 
	tabulIn_d[i-1 + jPro] + 
	tabulIn_d[i   + jPro] + 
	tabulIn_d[i+1 + jPro];
      if (tabulIn_d[i + jCor] && vizviv < 2) 
	tabulOut_d[i + jCor] = 0;
      else if (tabulIn_d[i + jCor] && vizviv > 3) 
	tabulOut_d[i + jCor] = 0;
      else if (!tabulIn_d[i + jCor] && vizviv == 3) 
	tabulOut_d[i + jCor] = 1;
      else
	tabulOut_d[i + jCor] = tabulIn_d[i + jCor];
    }
  }
}


// InitTabuleiros: Inicializa dois tabuleiros:
//                tabulIn com um veleiro 
//                tabulOut com celulas mortas

void InitTabul(int* tabulIn, int* tabulOut, int tam){
  int ij;

  for (ij=0; ij<(tam+2)*(tam+2); ij++) {
    tabulIn[ij] = 0;
    tabulOut[ij] = 0;
  }

  tabulIn[1*(tam+2)+3] = 1; 
  tabulIn[2*(tam+2)+1] = 1; 
  tabulIn[2*(tam+2)+3] = 1; 
  tabulIn[3*(tam+2)+2] = 1; 
  tabulIn[3*(tam+2)+3] = 1; 
}


// DumpTabuleiro: Imprime trecho do tabuleiro entre
//                as posicoes (pri,pri) e (ult,ult)
//                X representa celula viva
//                . representa celula morta

void DumpTabul(int * tabul, int tam, int first, int last, char* msg){
  int i, ij;

  printf("%s; trecho [%d:%d, %d:%d] do tabuleiro\n", msg, first, last, first, last);
  for (i=first; i<=last; i++) printf("="); printf("=\n");

  for (i=first; i<=last; i++) {
    for (ij=first*(tam+2)+i; ij<=last*(tam+2)+i; ij+=(tam+2))
      printf("%c", tabul[ij]? 'X' : '.');
    printf("\n");
  }
  for (i=first; i<=last; i++) printf("="); printf("=\n");
}


// Correto: Verifica se a configuracao final do tabuleiro
//          estah correta ou nao (veleiro no canto inferior esquerdo)

int Correto(int* tabul, int tam){
  int ij, ini, cnt;

  cnt = 0;
  for (ij=0; ij<(tam+2)*(tam+2); ij++) {
    cnt = cnt + tabul[ij];
  }

  ini = (tam-3)*(tam+2) + (tam-3);
  if (cnt == 5 &&
      tabul[1*(tam+2)+3+ini] &&
      tabul[2*(tam+2)+1+ini] &&
      tabul[2*(tam+2)+3+ini] &&
      tabul[3*(tam+2)+2+ini] &&
      tabul[3*(tam+2)+3+ini] )
    return(1);
  else 
    return(0);
}


// Trafega um veleiro de tamanho tam ao longo do tabuleiro
// Tamanho tam eh argumento de execucao

int main(int argc, char *argv[]) {
  
  #define MinTam 4
  // Declara��o de Variaveis na CPU (host)
  int i;
  int tam, tamBlk, nBlk;
  int* tabulIn_h;
  int* tabulOut_h;
  size_t size;
  double t00, t01, t02, t03;// Tempo na CPU
  double t10, t11, t12, t13;//Tempo na GPU
  
  //Declara��o de Variaveis da GPU (device)
  int* tabulIn_d;
  int* tabulOut_d;
  
  // obtem tamanho do tabuleiro
  if (argc != 3) {
    printf(" uso: <exec> <celulas por bloco> <quantos blocos>\n");
    exit(-1);
  }
  tamBlk = atoi(argv[1]);
  nBlk = atoi(argv[2]);
  tam = nBlk*tamBlk;
  size = (tam+2)*(tam+2)*sizeof(int);
  
  // tamanho minimo
  if (tam < MinTam) {
    printf("**ERRO** tamanho %d menor que o minimo %d\n", tam, MinTam);
    exit(-1);
  }

//**************************************************************************
// CPU
//**************************************************************************

  // aloca e inicializa tabuleiros na CPU
  t00 = wall_time();
  tabulIn_h  = (int *) malloc (size);
  tabulOut_h = (int *) malloc (size);
  InitTabul(tabulIn_h, tabulOut_h, tam);
  
  // dump tabuleiro inicial na CPU
  DumpTabul(tabulIn_h, tam, 1, 4, "Inicial::CPU");
  
  // avanca geracoes na CPU
  t01 = wall_time();
  for (i=0; i<2*(tam-3); i++) {
    UmaVida (tabulIn_h, tabulOut_h, tam);
    UmaVida (tabulOut_h, tabulIn_h, tam);
  }
  t02 = wall_time();
  
  // dump tabuleiro final na CPU
  DumpTabul(tabulIn_h, tam, tam-3, tam, "Final::CPU");
  
  // Correcao na CPU
  if (Correto(tabulIn_h, tam)) 
    printf("**RESULTADO CORRETO NA CPU**\n");
  else
    printf("**RESULTADO ERRADO NA CPU**\n");

  t03 = wall_time();

  printf("CPU tam=%d; tempos: init=%f, comp=%f, fim=%f, tot=%f \n", 
	 tam, t01-t00, t02-t01, t03-t02, t03-t00);//Printa resultados CPU

//**************************************************************************
// GPU
//**************************************************************************
	 
	// aloca tabuleiros na GPU e envia tabuleiros inicializados
	t10 = wall_time();
	hipMalloc((void**) &tabulIn_d, size);//Aloca na GPU
	hipMalloc((void**) &tabulOut_d, size);//Aloca na GPU
	InitTabul(tabulIn_h, tabulOut_h, tam);//Inicializa novamente para enviar tabuleiro
	hipMemcpy (tabulIn_d, tabulIn_h, size, hipMemcpyHostToDevice);//Envia para GPU
	hipMemcpy (tabulOut_d, tabulOut_h, size, hipMemcpyHostToDevice);//Envia para GPU

	// avanca geracoes na GPU
	t11 = wall_time();
	dim3 dB (tamBlk, tamBlk);//dimensionamento
	dim3 dG (nBlk, nBlk);//dimensionamento
	for (i=0; i<2*(tam-3); i++) {
		UmaVidaGPU<<< dG, dB >>>(tabulIn_d, tabulOut_d, tam);//chamada do kernel
		hipDeviceSynchronize();//sincroniza��o
		UmaVidaGPU<<< dG, dB >>>(tabulOut_d, tabulIn_d, tam);//chamada do kernel
		hipDeviceSynchronize();//outra sincroniza��o
	}
	t12 = wall_time();
	 
	// Recebe tabuleiros e dump tabuleiro final da GPU
	hipMemcpy (tabulIn_h, tabulIn_d, size, hipMemcpyDeviceToHost);
	hipMemcpy (tabulOut_h, tabulOut_d, size, hipMemcpyDeviceToHost);
	DumpTabul(tabulIn_h, tam, tam-3, tam, "Final::GPU"); //dump do tabuleiro da CPU reescrito pelo resultado da GPU

	// Correcao na GPU
	if (Correto(tabulIn_h, tam)) //verifica na memoria da CPU
		printf("**RESULTADO CORRETO NA GPU**\n");
	else
		printf("**RESULTADO ERRADO NA GPU**\n");

	t13 = wall_time();
	 
  printf("GPU tam=%d; tempos: init=%f, comp=%f, fim=%f, tot=%f \n", 
	 tam, t11-t10, t12-t11, t13-t12, t13-t10);//Printa resultados GPU

  //Desaloca mem�ria na CPU
  free(tabulIn_h);
  free(tabulOut_h);
  
  //Desaloca mem�ria na GPU
  hipFree(tabulIn_d);
  hipFree(tabulOut_d);

  exit(0);    
}
